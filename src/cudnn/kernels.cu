#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <exception>

#include "context.hpp"
#include "kernels.hpp"

using namespace upstride;

/**
 * @brief CUDA kernel cropping an input NCHW tensor along H and W dimensions
 * The output tensor is smaller or equal in size than the input tensor.
 * @param in            pointer to input values
 * @param out           pointer to output values
 * @param dx            horizontal shift
 * @param dy            vertical shift
 * @param inWidth       input tensor width
 * @param inHeight      input tensor height
 * @param outWidth      output tensor width
 * @param outHeight     output tensor height
 * @param depth         the depth of both input and output tensors (N times C times the element size)
 */
template <typename T>
__global__ void cropNCHW(const T* in, T* out, int dx, int dy, int inWidth, int inHeight, int outWidth, int outHeight, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < outWidth && y < outHeight && z < depth)
        out[(z * outHeight + y) * outWidth + x] = in[(z * inHeight + y + dy) * inWidth + x + dx];
}

/**
 * @brief CUDA kernel inserting an input NCHW tensor into an output NCHW tensor
 * The input tensor is smaller or equal in size than the output tensor.
 * @param in            pointer to input values
 * @param out           pointer to output values
 * @param dx            horizontal shift
 * @param dy            vertical shift
 * @param inWidth       input tensor width
 * @param inHeight      input tensor height
 * @param outWidth      output tensor width
 * @param outHeight     output tensor height
 * @param depth         the depth of both input and output tensors (N times C times the element size)
 */
template <typename T>
__global__ void insertNCHW(const T* in, T* out, int dx, int dy, int inWidth, int inHeight, int outWidth, int outHeight, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < inWidth && y < inHeight && z < depth)
        out[(z * outHeight + y + dy) * outWidth + x + dx] = in[(z * inHeight + y) * inWidth + x];
}

/**
 * @brief Rounding up integer division
 * 
 * @param n nominator
 * @param d denominator
 * @return closest integer greater than n/d 
 */
inline int ceili(int n, int d) {
    return (n + d - 1) / d;
}

/**
 * @brief Sets up a naive CUDA kernel grid config for a pointwise operation
 * @param shape         shape of the threads space to sample
 * @param dataFormat    data format of the corresponding shape
 * @param threads       number of threads per block (output)
 * @param blocks        number of thread blocks (ouptut)
 * @param numThreads    maximum number of threads per block
 */
inline static void makeGridConfig(const Shape& shape, DataFormat dataFormat, dim3& threads, dim3& blocks, const int numThreads = 64) {
    const int depth = shape.depth(dataFormat) * shape[0];
    const int z = std::min(numThreads, depth);
    const int xy = (int)std::sqrt(numThreads / z);
    threads = dim3(xy, xy, z);
    blocks = dim3(
        ceili(shape.width(dataFormat), threads.x),
        ceili(shape.height(dataFormat), threads.y),
        ceili(shape.depth(dataFormat) * shape[0], threads.z));
}

void upstride::cudnn::crop(const Tensor<const float>& input, Tensor<float>& output, DataFormat dataFormat, const IntPair& offset) {
    // check stuff
    const Shape& inShape = input.getShape();
    const Shape& outShape = output.getShape();

    if (dataFormat != DataFormat::NCHW)
        throw std::runtime_error("Unsupported data format");
    if (inShape.getSize() != 4 || outShape.getSize() != 4)
        throw std::runtime_error("Expecting four-dimenisonal input and output tensors");
    if (outShape.width(dataFormat) + offset.x < inShape.width(dataFormat) ||
        outShape.height(dataFormat) + offset.y < inShape.height(dataFormat))
        throw std::runtime_error("Cannot fit output tensor into input tensor");
    if (inShape.depth(dataFormat) != outShape.depth(dataFormat))
        throw std::runtime_error("Input / output depth mismatch");
    if (inShape[0] != outShape[0])
        throw std::runtime_error("Input / output batch size mismatch");

    // setting up thread grid
    dim3 threads, blocks;
    makeGridConfig(inShape, dataFormat, threads, blocks);

    // launching the kernel
    cropNCHW<<<blocks, threads>>>(
        input.getDataPtr(),
        output.getDataPtr(),
        offset.x, offset.y,
        inShape.width(dataFormat), inShape.height(dataFormat),
        outShape.width(dataFormat), outShape.height(dataFormat),
        inShape.depth(dataFormat) * inShape[0]);

    Context::raiseIfError();
}

void upstride::cudnn::insert(const Tensor<const float>& input, Tensor<float>& output, DataFormat dataFormat, const IntPair& offset) {
    // check stuff
    const Shape& inShape = input.getShape();
    const Shape& outShape = output.getShape();

    if (dataFormat != DataFormat::NCHW)
        throw std::runtime_error("Unsupported data format");
    if (inShape.getSize() != 4 || outShape.getSize() != 4)
        throw std::runtime_error("Expecting four-dimenisonal input and output tensors");
    if (inShape.width(dataFormat) + offset.x > outShape.width(dataFormat) ||
        inShape.height(dataFormat) + offset.y > outShape.height(dataFormat))
        throw std::runtime_error("Cannot fit input tensor into output tensor");
    if (inShape.depth(dataFormat) != outShape.depth(dataFormat))
        throw std::runtime_error("Input / output depth mismatch");
    if (inShape[0] != outShape[0])
        throw std::runtime_error("Input / output batch size mismatch");

    // setting up thread grid
    dim3 threads, blocks;
    makeGridConfig(outShape, dataFormat, threads, blocks);

    // launching the kernel
    insertNCHW<<<blocks, threads>>>(
        input.getDataPtr(),
        output.getDataPtr(),
        offset.x, offset.y,
        inShape.width(dataFormat), inShape.height(dataFormat),
        outShape.width(dataFormat), outShape.height(dataFormat),
        outShape.depth(dataFormat) * outShape[0]);

    Context::raiseIfError();
}