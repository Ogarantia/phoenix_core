#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdexcept>

#include "context.hpp"
#include "kernels.hpp"

using namespace upstride;

static const int NUM_THREADS = 1024;  //!< default number of CUDA threads per block

template <typename T>
__global__ void accumulateAdd(T* acc, const T* term, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length)
        acc[i] += term[i];
}

template <typename T>
__global__ void accumulateSub(T* acc, const T* term, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length)
        acc[i] -= term[i];
}

/**
 * @brief CUDA kernel cropping an input NCHW tensor along H and W dimensions
 * The output tensor is smaller or equal in size than the input tensor.
 * @param in            pointer to input values
 * @param out           pointer to output values
 * @param dx            horizontal shift
 * @param dy            vertical shift
 * @param inWidth       input tensor width
 * @param inHeight      input tensor height
 * @param outWidth      output tensor width
 * @param outHeight     output tensor height
 * @param depth         the depth of both input and output tensors (N times C times the element size)
 */
template <typename T>
__global__ void cropNCHW(const T* in, T* out, int dx, int dy, int inWidth, int inHeight, int outWidth, int outHeight, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < outWidth && y < outHeight && z < depth)
        out[(z * outHeight + y) * outWidth + x] = in[(z * inHeight + y + dy) * inWidth + x + dx];
}

/**
 * @brief CUDA kernel inserting an input NCHW tensor into an output NCHW tensor
 * The input tensor is smaller or equal in size than the output tensor.
 * @param in            pointer to input values
 * @param out           pointer to output values
 * @param dx            horizontal shift
 * @param dy            vertical shift
 * @param inWidth       input tensor width
 * @param inHeight      input tensor height
 * @param outWidth      output tensor width
 * @param outHeight     output tensor height
 * @param depth         the depth of both input and output tensors (N times C times the element size)
 */
template <typename T>
__global__ void insertNCHW(const T* in, T* out, int dx, int dy, int inWidth, int inHeight, int outWidth, int outHeight, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < inWidth && y < inHeight && z < depth)
        out[(z * outHeight + y + dy) * outWidth + x + dx] = in[(z * inHeight + y) * inWidth + x];
}

template <typename T>
__global__ void addBiasNCHW(T* tensor, const T* bias, int width, int height, int depth, int batchSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < width && y < height && z < depth)
        for (int n = 0; n < batchSize; ++n)
            tensor[((n * depth + z) * height + y) * width + x] += bias[z];
}

template <typename T>
__global__ void addBiasNC(T* tensor, const T* bias, int length, int batchSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < length)
        for (int n = 0; n < batchSize; ++n)
            tensor[n * length + x] += bias[x];
}


/**
 * @brief Sets up a simple CUDA kernel grid config for a pointwise operation
 * @param shape         shape of the threads space to sample
 * @param dataFormat    data format of the corresponding shape
 * @param threads       number of threads per block (output)
 * @param blocks        number of thread blocks (ouptut)
 * @param numThreads    maximum number of threads per block
 */
inline static void makeGridConfig(const Shape& shape, DataFormat dataFormat, dim3& threads, dim3& blocks, const int numThreads = NUM_THREADS) {
    const int depth = shape.depth(dataFormat) * shape[0];
    const int z = std::min(cudnn::Context::MAX_BLOCK_DEPTH, depth);
    const int xy = (int)std::sqrt(numThreads / z);
    threads = dim3(xy, xy, z);
    blocks = dim3(
        ceili(shape.width(dataFormat), threads.x),
        ceili(shape.height(dataFormat), threads.y),
        ceili(depth, threads.z));
}

template <typename T>
void crop(const Tensor<device::CUDA, T>& input, Tensor<device::CUDA, T>& output, DataFormat dataFormat, const IntPair& offset) {
    // check stuff
    const Shape& inShape = input.getShape();
    const Shape& outShape = output.getShape();

    if (dataFormat != DataFormat::NCHW)
        throw std::runtime_error("Unsupported data format");
    if (inShape.getSize() != 4 || outShape.getSize() != 4)
        throw std::runtime_error("Expecting four-dimenisonal input and output tensors");
    if (outShape.width(dataFormat) + offset.x < inShape.width(dataFormat) ||
        outShape.height(dataFormat) + offset.y < inShape.height(dataFormat))
        throw std::runtime_error("Cannot fit output tensor into input tensor");
    if (inShape.depth(dataFormat) != outShape.depth(dataFormat))
        throw std::runtime_error("Input / output depth mismatch");
    if (inShape[0] != outShape[0])
        throw std::runtime_error("Input / output batch size mismatch");

    // setting up thread grid
    dim3 threads, blocks;
    makeGridConfig(inShape, dataFormat, threads, blocks);

    // launching the kernel
    cropNCHW<<<blocks, threads, 0, input.getDevice().stream()>>>(
        input.getDataPtr(),
        output.getDataPtr(),
        offset.x, offset.y,
        inShape.width(dataFormat), inShape.height(dataFormat),
        outShape.width(dataFormat), outShape.height(dataFormat),
        inShape.depth(dataFormat) * inShape[0]);

    cudnn::Context::raiseIfError();
}

template<typename T>
void addBias(Tensor<device::CUDA, T>& tensor, const Tensor<device::CUDA, const T>& bias, DataFormat dataFormat) {
    if (dataFormat != DataFormat::NCHW && dataFormat != DataFormat::NC)
        throw std::runtime_error("Unsupported data format");
    const Shape& shape = tensor.getShape();
    if (dataFormat == DataFormat::NCHW && shape.getSize() != 4)
        throw std::runtime_error("Expecting a four-dimensional tensor");
    if (dataFormat == DataFormat::NC && shape.getSize() != 2)
        throw std::runtime_error("Expecting a two-dimensional tensor");
    if (shape.depth(dataFormat) != bias.getShape().numel())
        throw std::runtime_error("Tensor and bias sizes mismatch");

    if (dataFormat == DataFormat::NCHW) {
        dim3 threads, blocks;
        makeGridConfig(shape, dataFormat, threads, blocks);
        addBiasNCHW<<<blocks, threads, 0, tensor.getDevice().stream()>>>(
            tensor.getDataPtr(), bias.getDataPtr(),
            shape.width(dataFormat), shape.height(dataFormat), shape.depth(dataFormat), shape[0]);
    }
    else if (dataFormat == DataFormat::NC) {
        const int length = shape.depth(dataFormat);
        addBiasNC<<<ceili(shape[1], NUM_THREADS), NUM_THREADS, 0, tensor.getDevice().stream()>>>(
            tensor.getDataPtr(), bias.getDataPtr(), shape[1], shape[0]);
    }
    else
        throw std::runtime_error("addBias is currently not implemented for the given dataFormat.");
}

template <typename T>
void insert(const Tensor<device::CUDA, const T>& input, Tensor<device::CUDA, T>& output, DataFormat dataFormat, const IntPair& offset) {
    // check stuff
    const Shape& inShape = input.getShape();
    const Shape& outShape = output.getShape();

    if (dataFormat != DataFormat::NCHW)
        throw std::runtime_error("Unsupported data format");
    if (inShape.getSize() != 4 || outShape.getSize() != 4)
        throw std::runtime_error("Expecting four-dimenisonal input and output tensors");
    if (inShape.width(dataFormat) + offset.x > outShape.width(dataFormat) ||
        inShape.height(dataFormat) + offset.y > outShape.height(dataFormat))
        throw std::runtime_error("Cannot fit input tensor into output tensor");
    if (inShape.depth(dataFormat) != outShape.depth(dataFormat))
        throw std::runtime_error("Input / output depth mismatch");
    if (inShape[0] != outShape[0])
        throw std::runtime_error("Input / output batch size mismatch");

    // setting up thread grid
    dim3 threads, blocks;
    makeGridConfig(outShape, dataFormat, threads, blocks);

    // launching the kernel
    insertNCHW<<<blocks, threads, 0, input.getDevice().stream()>>>(
        input.getDataPtr(),
        output.getDataPtr(),
        offset.x, offset.y,
        inShape.width(dataFormat), inShape.height(dataFormat),
        outShape.width(dataFormat), outShape.height(dataFormat),
        outShape.depth(dataFormat) * outShape[0]);

    cudnn::Context::raiseIfError();
}

namespace upstride {
namespace cudnn {

template <>
void crop(const Tensor<device::CUDA, float>& input, Tensor<device::CUDA, float>& output, DataFormat dataFormat, const IntPair& offset) {
    ::crop(input, output, dataFormat, offset);
}

template <>
void insert(const Tensor<device::CUDA, const float>& input, Tensor<device::CUDA, float>& output, DataFormat dataFormat, const IntPair& offset) {
    ::insert(input, output, dataFormat, offset);
}

template<>
void addBias(Tensor<device::CUDA, float>& tensor, const Tensor<device::CUDA, const float>& bias, DataFormat dataFormat) {
    ::addBias(tensor, bias, dataFormat);
}

template <>
void accumulateAdd(const device::CUDA& device, float* accumulator, const float* term, int length) {
    ::accumulateAdd<<<ceili(length, NUM_THREADS), NUM_THREADS, 0, device.stream()>>>(accumulator, term, length);
}

template <>
void accumulateSub(const device::CUDA& device, float* accumulator, const float* term, int length) {
    ::accumulateSub<<<ceili(length, NUM_THREADS), NUM_THREADS, 0, device.stream()>>>(accumulator, term, length);
}

#ifdef UPSTRIDE_ENABLE_FP16
template <>
void crop(const Tensor<device::CUDA, half>& input, Tensor<device::CUDA, half>& output, DataFormat dataFormat, const IntPair& offset) {
    ::crop(input, output, dataFormat, offset);
}

template <>
void insert(const Tensor<device::CUDA, const half>& input, Tensor<device::CUDA, half>& output, DataFormat dataFormat, const IntPair& offset) {
    ::insert(input, output, dataFormat, offset);
}

template<>
void addBias(Tensor<device::CUDA, half>& tensor, const Tensor<device::CUDA, const half>& bias, DataFormat dataFormat) {
    ::addBias(tensor, bias, dataFormat);
}

template <>
void accumulateAdd(const device::CUDA& device, half* accumulator, const half* term, int length) {
    ::accumulateAdd<<<ceili(length, NUM_THREADS), NUM_THREADS, 0, device.stream()>>>(accumulator, term, length);
    cudnn::Context::raiseIfError();
}

template <>
void accumulateSub(const device::CUDA& device, half* accumulator, const half* term, int length) {
    ::accumulateSub<<<ceili(length, NUM_THREADS), NUM_THREADS, 0, device.stream()>>>(accumulator, term, length);
    cudnn::Context::raiseIfError();
}
#endif

}  // namespace cudnn

}  // namespace upstride